#include "hip/hip_runtime.h"

#include "wrapper_helpers.h"
#include "util/cutil.cuh"
#include "util/reduction.cuh"
#include "hip/hip_runtime.h"

static cuda_time *_total;
static cuda_time *_kernel;
static cuda_time *_reduce;


__device__ int warmup_memory = 0;
__global__ void warmup_kernel(){ warmup_memory ^= 1; }

void wrapper_setup(
	search_parameters p, char **d_text, char **d_pattern, int **d_match)
{
	gpuErrchk( hipMalloc((void**)d_text, 	 p.text_size * sizeof(char)) );
	gpuErrchk( hipMalloc((void**)d_match,   p.text_size * sizeof(int)) );
	gpuErrchk( hipMalloc((void**)d_pattern, p.pattern_size * sizeof(char)) );
	
    // Warmup kernel to filter out startup overhead
    warmup_kernel<<<1, 1>>>();

	gpuErrchk( hipEventRecord(_total->start) );

	gpuErrchk( hipMemcpy(*d_text, p.text, 
		p.text_size * sizeof(char), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemset(*d_match, 0, p.text_size * sizeof(int)) );
	gpuErrchk( hipMemcpy(*d_pattern, p.pattern, 
		p.pattern_size * sizeof(char), hipMemcpyHostToDevice) );
}

void wrapper_teardown(
	search_parameters p, search_info *timers, 
	char *d_text, char *d_pattern, int *d_match)
{
	int* d_match_count;
	if (p.gpu_reduction){
		gpuErrchk( hipMalloc((void**)&d_match_count, sizeof(int)) );
		gpuErrchk( hipEventRecord(_reduce->start) );
		device_reduce_block_atomic(d_match, d_match_count, p.text_size);
		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipMemcpy(p.match, d_match_count, sizeof(int), hipMemcpyDeviceToHost) );
		gpuErrchk( hipEventRecord(_reduce->stop) );
	}
	else {
		gpuErrchk( hipMemcpy(p.match, d_match, 
			p.text_size * sizeof(int), hipMemcpyDeviceToHost) );
	}

	gpuErrchk( hipEventRecord(_total->stop) );

	gpuErrchk( hipFree(d_text) );
	gpuErrchk( hipFree(d_pattern) );
	gpuErrchk( hipFree(d_match) );
	if (p.gpu_reduction){
		gpuErrchk( hipFree(d_match_count) );
	}

	gpuErrchk( hipEventSynchronize(_total->stop) );
	gpuErrchk( hipEventElapsedTime(
		&(timers->kernel_duration), _kernel->start, _kernel->stop) );
	gpuErrchk( hipEventElapsedTime(
		&(timers->total_duration), _total->start, _total->stop) );
	if (p.gpu_reduction){
		gpuErrchk( hipEventElapsedTime(
			&(timers->reduce_duration), _reduce->start, _reduce->stop) );
	}

	gpuErrchk( hipEventDestroy(_total->start) );
	gpuErrchk( hipEventDestroy(_total->stop) );
	gpuErrchk( hipEventDestroy(_kernel->start) );
	gpuErrchk( hipEventDestroy(_kernel->stop) );
	gpuErrchk( hipEventDestroy(_reduce->start) );
	gpuErrchk( hipEventDestroy(_reduce->stop) );

	free(_reduce);
}

void get_kernel_configuration(
	search_parameters p, unsigned int *grid_dim, unsigned int *block_dim)
{
	get_kernel_configuration_shared(p, 0, grid_dim, block_dim);
}

void get_kernel_configuration_shared(
	search_parameters p, int shared_size,
	unsigned int *grid_dim, unsigned int *block_dim)
{
	int block_count = divUp(p.text_size, p.stride_length);

	if (shared_size != 0){
		int block_dim_max = 
			((shared_size - p.pattern_size) / p.stride_length / 32) * 32;
		if (block_dim_max > block_count)
			*block_dim = min( block_count, p.block_dim );
		else
			*block_dim = min( block_dim_max, p.block_dim );
	}
	else
		*block_dim = block_count > p.block_dim ?
		 p.block_dim : divUp(block_count, 32) * 32;


	*grid_dim = divUp(p.text_size, (*block_dim) * p.stride_length);
}

void setup_timers(cuda_time *kernel, cuda_time *total)
{
	_total = total;
	_kernel = kernel;
	_reduce = (struct cuda_time*)malloc(sizeof(struct cuda_time));
	gpuErrchk( hipEventCreate(&(_total->start)) );
	gpuErrchk( hipEventCreate(&(_total->stop)) );
	gpuErrchk( hipEventCreate(&(_kernel->start)) );
	gpuErrchk( hipEventCreate(&(_kernel->stop)) );
	gpuErrchk( hipEventCreate(&(_reduce->start)) );
	gpuErrchk( hipEventCreate(&(_reduce->stop)) );
}
